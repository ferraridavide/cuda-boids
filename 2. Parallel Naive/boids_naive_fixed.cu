#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#define CANVAS_SIZE 200.0f
#define NEIGHBOR_RADIUS 5.0f
#define SEPARATION_RADIUS 2.0f
#define MAX_SPEED 20.0f
#define MAX_FORCE 50.0f
#define ALIGNMENT 1.0f
#define COHESION 0.5f
#define SEPARATION 1.5f
#define DT 0.01f

__device__ __host__ inline float2 f2(float x, float y)
{
    float2 v{x, y};
    return v;
}

// Inline: replace this function call with its code body -> Less overhead
__device__ __host__ inline float2 f2add(float2 a, float2 b) { return f2(a.x + b.x, a.y + b.y); }
__device__ __host__ inline float2 f2sub(float2 a, float2 b) { return f2(a.x - b.x, a.y - b.y); }
__device__ __host__ inline float2 f2mul(float2 a, float s) { return f2(a.x * s, a.y * s); }
__device__ __host__ inline float f2dot(float2 a, float2 b) { return a.x * b.x + a.y * b.y; }
__device__ __host__ inline float f2len(float2 a) { return sqrtf(f2dot(a, a)); }
__device__ inline float2 clamp_vector(float2 v, float max)
{
    float len2 = f2dot(v, v);
    float max2 = max * max;
    if (len2 > max2)
    {
        float invL = rsqrtf(len2);
        v = f2mul(v, max * invL);
    }
    return v;
}

// Wrap coordinate into [0, CANVAS_SIZE) 
__device__ inline float warp_coord(float coord)
{
    float r = fmodf(coord, CANVAS_SIZE);
    if (r < 0.0f)
        r += CANVAS_SIZE;
    return r;
}

// // Wrap coordinates around toroidal space [0, CANVAS_SIZE)
__device__ inline float2 min_image(float2 r)
{
    const float half = CANVAS_SIZE * 0.5f;
    if (r.x > half)
        r.x -= CANVAS_SIZE;
    if (r.x < -half)
        r.x += CANVAS_SIZE;
    if (r.y > half)
        r.y -= CANVAS_SIZE;
    if (r.y < -half)
        r.y += CANVAS_SIZE;
    return r;
}

__global__ void update_boids(
    const float2 *pos_in,
    const float2 *vel_in,
    float2 *pos_out,
    float2 *vel_out,
    int boids)
{
    int myself = blockIdx.x * blockDim.x + threadIdx.x;
    if (myself >= boids)
        return;

    float2 my_pos = pos_in[myself];
    float2 my_vel = vel_in[myself];

    float2 sum_vel = f2(0.0f, 0.0f);
    float2 sum_pos = f2(0.0f, 0.0f);
    float2 sum_sep = f2(0.0f, 0.0f);
    int countAlign = 0;
    int countCoh = 0;

    const float neighR2 = NEIGHBOR_RADIUS * NEIGHBOR_RADIUS;
    const float sepR2 = SEPARATION_RADIUS * SEPARATION_RADIUS;

    for (int other = 0; other < boids; ++other)
    {
        if (other == myself)
            continue;

        float2 r = f2sub(pos_in[other], my_pos);
        r = min_image(r);
        float d2 = f2dot(r, r);

        if (d2 < neighR2)
        {
            sum_vel = f2add(sum_vel, vel_in[other]);
            sum_pos = f2add(sum_pos, f2add(pos_in[myself], r));
            countAlign++;
            countCoh++;
        }

        if (d2 < sepR2)
        {
            float invDist = rsqrtf(d2 + 1e-6f);
            float2 away = f2mul(r, -invDist); // -r/|r|
            sum_sep = f2add(sum_sep, away);
        }
    }
    // Alignment
    float2 align = f2(0, 0);
    if (countAlign > 0)
    {
        align = f2mul(sum_vel, 1.0f / (float)countAlign);
        align = f2sub(align, my_vel);
    }
    // Cohesion
    float2 coh = f2(0, 0);
    if (countCoh > 0)
    {
        float2 center = f2mul(sum_pos, 1.0f / (float)countCoh);
        coh = min_image(f2sub(center, my_pos));
    }
    // Combine forces
    float2 accel = f2(0, 0);
    accel = f2add(accel, f2mul(align, ALIGNMENT));
    accel = f2add(accel, f2mul(coh, COHESION));
    accel = f2add(accel, f2mul(sum_sep, SEPARATION));
    accel = clamp_vector(accel, MAX_FORCE);
    // Integrate 
    float2 new_vel = f2add(my_vel, f2mul(accel, DT));
    new_vel = clamp_vector(new_vel, MAX_SPEED);
    float2 new_pos = f2add(my_pos, f2mul(new_vel, DT));

    new_pos.x = warp_coord(new_pos.x);
    new_pos.y = warp_coord(new_pos.y);

    pos_out[myself] = new_pos;
    vel_out[myself] = new_vel;
}

static inline float frand() { return (float)rand() / (float)RAND_MAX; }
static void initialize_boids(float2 *pos, float2 *vel, int count)
{
    srand(1234);
    for (int i = 0; i < count; ++i)
    {
        pos[i] = f2(frand() * CANVAS_SIZE, frand() * CANVAS_SIZE);

        float angle = frand() * (float)(2.0 * M_PI);
        float intensity = frand() * MAX_SPEED;

        vel[i] = f2(intensity * cosf(angle), intensity * sinf(angle));
    }
}
int main(int argc, char **argv)
{
    int boids = 2048; // default
    int steps = 100;
    int TPB = 1024;

    if (argc > 1)
        boids = atoi(argv[1]);
    if (argc > 2)
        steps = atoi(argv[2]);

    size_t bytes = (size_t)boids * sizeof(float2);
    // Host buffers
    float2 *h_pos = (float2 *)malloc(bytes);
    float2 *h_vel = (float2 *)malloc(bytes);

    initialize_boids(h_pos, h_vel, boids);
    // Device buffers (double-buffered)
    float2 *d_pos_in, *d_vel_in;
    float2 *d_pos_out, *d_vel_out;
    hipMalloc((void **)&d_pos_in, bytes);
    hipMalloc((void **)&d_vel_in, bytes);
    hipMalloc((void **)&d_pos_out, bytes);
    hipMalloc((void **)&d_vel_out, bytes);
    hipMemcpy(d_pos_in, h_pos, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_vel_in, h_vel, bytes, hipMemcpyHostToDevice);
    dim3 block(TPB);
    dim3 grid((boids + TPB - 1) / TPB);
    // timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int s = 0; s < steps; ++s)
    {
        update_boids<<<grid, block>>>(d_pos_in, d_vel_in, d_pos_out, d_vel_out, boids);
        // Swap buffers
        float2 *tmp_pos = d_pos_in;
        d_pos_in = d_pos_out;
        d_pos_out = tmp_pos;
        float2 *tmp_vel = d_vel_in;
        d_vel_in = d_vel_out;
        d_vel_out = tmp_vel;
    }
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    // Copy back final state
    hipMemcpy(h_pos, d_pos_in, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_vel, d_vel_in, bytes, hipMemcpyDeviceToHost);
    // printf("%.3f",ms);
    printf("Naive 2D boids (double-buffered): %d boids × %d steps in %.3f ms (%.3f ms/step)\n",
            boids, steps, ms, ms / (float)steps);
    FILE *fp = fopen("boids_naive.csv", "w");
    if (!fp)
    {
        fprintf(stderr, "Failed to open boids_naive.csv for writing\n");
    }
    else
    {
        fprintf(fp, "id,x,y,dir_x,dir_y\n");
        for (int i = 0; i < boids; ++i)
        {
            float x = h_pos[i].x;
            float y = h_pos[i].y;
            float vx = h_vel[i].x;
            float vy = h_vel[i].y;
            float speed = f2len(f2(vx, vy));
            float dx = 0.0f, dy = 0.0f;
            if (speed > 1e-6f)
            {
                float invSpeed = 1.0f / speed;
                dx = vx * invSpeed;
                dy = vy * invSpeed;
            }
            fprintf(fp, "%d,%.6f,%.6f,%.6f,%.6f\n", i, x, y, dx, dy);
        }
        fclose(fp);
        printf("Wrote boids_naive.csv\n");
    }
    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_pos_in);
    hipFree(d_vel_in);
    hipFree(d_pos_out);
    hipFree(d_vel_out);
    free(h_pos);
    free(h_vel);
    return 0;

}
